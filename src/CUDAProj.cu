#include "hip/hip_runtime.h"
#include "Utils.h"
#include "CudaProj.h"
#include "common_utils.cuh"
// includes, system

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
/* Using updated (v2) interfaces to cublas and cusparse */
#include <hip/hip_runtime.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA SDK samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and intialization
#include "kernels.cuh"

void process(const char* name, int argc, char **argv) {
//nowe deklaracje
	int numSamples, aggHeapCount, aggHeapSize;
	float *h_samples, *h_aggr_min, *h_aggr_max, *h_aggr_avg;
	float *d_samples, *d_aggr_min, *d_aggr_max, *d_aggr_avg;
// This will pick the best possible CUDA capable device
	initCuda(argc, argv);
//allocate memory on cpu
	h_samples = ReadFile(name, &numSamples);

	aggHeapCount = getAggOffset(numSamples, AGG_ALL);
	aggHeapSize = aggHeapCount * sizeof(float);
	h_aggr_min = (float*) malloc(aggHeapSize);
	h_aggr_max = (float*) malloc(aggHeapSize);
	h_aggr_avg = (float*) malloc(aggHeapSize);
	cleanArray(aggHeapCount, h_aggr_min);
	cleanArray(aggHeapCount, h_aggr_max);
	cleanArray(aggHeapCount, h_aggr_avg);
#ifdef DEBUG
	printf("numSamples = %d\n", numSamples);
	printf("heapCount = %d\n", aggHeapCount);
	printf("heapSize = %d\n", aggHeapSize);

#endif
//allocate memory on gpu
	checkCudaErrors(hipMalloc((void ** ) &d_samples, numSamples * sizeof(float)));	//todo zrobić hipMalloc dla poszczególnych agregacji
	checkCudaErrors(hipMalloc((void ** ) &d_aggr_min, aggHeapSize));
	checkCudaErrors(hipMalloc((void ** ) &d_aggr_max, aggHeapSize));
	checkCudaErrors(hipMalloc((void ** ) &d_aggr_avg, aggHeapSize));
	//clean gpu arrays
	checkCudaErrors(hipMemcpy(d_aggr_min, h_aggr_min, aggHeapSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_aggr_max, h_aggr_max, aggHeapSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_aggr_avg, h_aggr_avg, aggHeapSize, hipMemcpyHostToDevice));

//transfer samples from cpu to gpu
	checkCudaErrors(hipMemcpy(d_samples, h_samples, numSamples * sizeof(float), hipMemcpyHostToDevice));

//	int threadsPerBlock = 512;
	int threadsPerBlock = AGG_TEST_18;	//TODO zoptymalizować ( AGG_TEST_18/AGG_TEST_3 )
//tworzymy tyle wątków ile potrzeba do policzenia najmniejszej agregacji
	int blocksPerGrid = divceil(getAggCount(numSamples, AGG_SAMPLE), threadsPerBlock);
	int cacheSize = threadsPerBlock * sizeof(float) * NUM_AGGREGATORS;	//every thread calculates AGG_SEC_10{min, max,avg}

#ifdef DEBUG
//	printf("host address device min: %p, max: %p, avg: %p\n", d_aggr_min, d_aggr_max, d_aggr_avg);
	printf("threadsPerBlock = %d, blocksPerGrid = %d, totalThreads = %d, sharedSize = %d\n", threadsPerBlock, blocksPerGrid, threadsPerBlock * blocksPerGrid, cacheSize);
#endif
	agg_kernel_1<<<blocksPerGrid, threadsPerBlock, cacheSize>>>(numSamples, d_samples, cacheSize, d_aggr_min, d_aggr_max, d_aggr_avg);
	//wywołanie kernela zbierającego dane z niezależnych bloków (zatem mamy tylko jeden blok)
	threadsPerBlock = blocksPerGrid;
	blocksPerGrid = 1;
	cacheSize = threadsPerBlock * sizeof(float) * NUM_AGGREGATORS;
	agg_kernel_2<<<blocksPerGrid, threadsPerBlock, cacheSize>>>(numSamples, cacheSize, d_aggr_min, d_aggr_max, d_aggr_avg);
	checkCudaErrors(hipMemcpy(h_aggr_min, d_aggr_min, aggHeapSize, hipMemcpyDeviceToHost));
//	printf("\nskopiowalem aggr min aggHeapCount: %d\n", aggHeapCount);
	checkCudaErrors(hipMemcpy(h_aggr_max, d_aggr_max, aggHeapSize, hipMemcpyDeviceToHost));
//	printf("\nskopiowalem aggr max\n");
	checkCudaErrors(hipMemcpy(h_aggr_avg, d_aggr_avg, aggHeapSize, hipMemcpyDeviceToHost));
//	printf("\nskopiowalem aggr avg\n");
	printHeap(numSamples, h_aggr_min);
	hipFree(d_samples);
	hipFree(d_aggr_min);
	hipFree(d_aggr_max);
	hipFree(d_aggr_avg);
	hipDeviceReset();
	printf("\n\n------------------ END ------------------\n");
	free(h_samples);
	free(h_aggr_min);
	free(h_aggr_max);
	free(h_aggr_avg);
}
int main(int argc, char **argv) {
//	int size = 20;
//	;
//	printf("offset of 1: %d\n", getAggOffset(size, AGG_TEST_1));
//	printf("offset of 3: %d\n", getAggOffset(size, AGG_TEST_3));
//	printf("offset of 6: %d\n", getAggOffset(size, AGG_TEST_6));
//	printf("offset of 18: %d\n", getAggOffset(size, AGG_TEST_18));
//	printf("offset of 36: %d\n", getAggOffset(size, AGG_TEST_36));
//	printf("offset of 108: %d\n", getAggOffset(size, AGG_TEST_108));
//	printf("heap count: %d\n", getAggOffset(size, AGG_ALL));
#ifdef TEST
//	process("Test_data.txt", argc, argv);
	process("data/Osoba_cut.txt", argc, argv);
#else
	process("data/Osoba_concat.txt", argc, argv);
#endif
	CHECK_LAUNCH_ERROR()
				;
}
